#include "hip/hip_runtime.h"
#include "spmv_cuda_crs_impl.cuh"

namespace SMAX::KERNELS::SPMV::CUDA {

template <typename IT, typename VT>
__global__ void
naive_crs_spmv_cuda(const ULL n_rows, const IT *SMAX_RESTRICT col,
                    const IT *SMAX_RESTRICT row_ptr,
                    const VT *SMAX_RESTRICT val, const VT *SMAX_RESTRICT x,
                    VT *SMAX_RESTRICT y) {

    ULL row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n_rows) {
        VT sum = VT{0};

        for (IT j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
            sum += val[j] * x[col[j]];
        }

        y[row] = sum;
    }
}

template <typename IT, typename VT>
void naive_crs_spmv_cuda_launcher(const ULL n_rows, const IT *SMAX_RESTRICT col,
                                  const IT *SMAX_RESTRICT row_ptr,
                                  const VT *SMAX_RESTRICT val,
                                  const VT *SMAX_RESTRICT x,
                                  VT *SMAX_RESTRICT y) {

    // CUDA_TPB selected at compile time
    ULL blocks = (n_rows + CUDA_TPB - 1) / CUDA_TPB;

    // clang-format off
    naive_crs_spmv_cuda<IT, VT><<<blocks, CUDA_TPB>>>(n_rows, col, row_ptr, val, x, y);
    // clang-format on

    // Synchronize device to ensure kernel execution completes
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in naive_crs_spmv_cuda_launcher: %s\n",
                hipGetErrorString(err));
        std::exit(EXIT_FAILURE); // or throw an exception depending on your
                                 // error model
    }
}

// // Macro for cuda kernel instantiation
// #define INSTANTIATE_CRS_SPMV_KERNEL(IndexType, ValueType) \
//     template __global__ void naive_crs_spmv_cuda<IndexType, ValueType>( \
//         const ULL, const IndexType *, const IndexType *, const ValueType *, \
//         const ValueType *, ValueType *);

// // Macro for launcher instantiation
// #define INSTANTIATE_CRS_SPMV_LAUNCHER(IndexType, ValueType) \
//     template void naive_crs_spmv_cuda_launcher<IndexType, ValueType>( \
//         const ULL, const IndexType *, const IndexType *, const ValueType *, \
//         const ValueType *, ValueType *);

// // Master macro to instantiate both
// #define INSTANTIATE_CRS_SPMV(IndexType, ValueType) \
//     INSTANTIATE_CRS_SPMV_KERNEL(IndexType, ValueType); \
//     INSTANTIATE_CRS_SPMV_LAUNCHER(IndexType, ValueType);

// #define INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(IndexType)                           \
//     INSTANTIATE_CRS_SPMV(IndexType, float);                                    \
//     INSTANTIATE_CRS_SPMV(IndexType, double);

// INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(int16_t);
// INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(int32_t);
// INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(int64_t);
// INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(uint16_t);
// INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(uint32_t);
// INSTANTIATE_CRS_SPMV_FLOAT_DOUBLE(uint64_t);

// clang-format off
template __global__ void naive_crs_spmv_cuda<int16_t, float>(ULL, const int16_t*, const int16_t*, const float*, const float*, float*);
template __global__ void naive_crs_spmv_cuda<int16_t, double>(ULL, const int16_t*, const int16_t*, const double*, const double*, double*);
template __global__ void naive_crs_spmv_cuda<int32_t, float>(ULL, const int32_t*, const int32_t*, const float*, const float*, float*);
template __global__ void naive_crs_spmv_cuda<int32_t, double>(ULL, const int32_t*, const int32_t*, const double*, const double*, double*);
template __global__ void naive_crs_spmv_cuda<int64_t, float>(ULL, const int64_t*, const int64_t*, const float*, const float*, float*);
template __global__ void naive_crs_spmv_cuda<int64_t, double>(ULL, const int64_t*, const int64_t*, const double*, const double*, double*);
template __global__ void naive_crs_spmv_cuda<uint16_t, float>(ULL, const uint16_t*, const uint16_t*, const float*, const float*, float*);
template __global__ void naive_crs_spmv_cuda<uint16_t, double>(ULL, const uint16_t*, const uint16_t*, const double*, const double*, double*);
template __global__ void naive_crs_spmv_cuda<uint32_t, float>(ULL, const uint32_t*, const uint32_t*, const float*, const float*, float*);
template __global__ void naive_crs_spmv_cuda<uint32_t, double>(ULL, const uint32_t*, const uint32_t*, const double*, const double*, double*);
template __global__ void naive_crs_spmv_cuda<uint64_t, float>(ULL, const uint64_t*, const uint64_t*, const float*, const float*, float*);
template __global__ void naive_crs_spmv_cuda<uint64_t, double>(ULL, const uint64_t*, const uint64_t*, const double*, const double*, double*);

template void naive_crs_spmv_cuda_launcher<int16_t, float>(ULL, const int16_t*, const int16_t*, const float*, const float*, float*);
template void naive_crs_spmv_cuda_launcher<int16_t, double>(ULL, const int16_t*, const int16_t*, const double*, const double*, double*);
template void naive_crs_spmv_cuda_launcher<int32_t, float>(ULL, const int32_t*, const int32_t*, const float*, const float*, float*);
template void naive_crs_spmv_cuda_launcher<int32_t, double>(ULL, const int32_t*, const int32_t*, const double*, const double*, double*);
template void naive_crs_spmv_cuda_launcher<int64_t, float>(ULL, const int64_t*, const int64_t*, const float*, const float*, float*);
template void naive_crs_spmv_cuda_launcher<int64_t, double>(ULL, const int64_t*, const int64_t*, const double*, const double*, double*);
template void naive_crs_spmv_cuda_launcher<uint16_t, float>(ULL, const uint16_t*, const uint16_t*, const float*, const float*, float*);
template void naive_crs_spmv_cuda_launcher<uint16_t, double>(ULL, const uint16_t*, const uint16_t*, const double*, const double*, double*);
template void naive_crs_spmv_cuda_launcher<uint32_t, float>(ULL, const uint32_t*, const uint32_t*, const float*, const float*, float*);
template void naive_crs_spmv_cuda_launcher<uint32_t, double>(ULL, const uint32_t*, const uint32_t*, const double*, const double*, double*);
template void naive_crs_spmv_cuda_launcher<uint64_t, float>(ULL, const uint64_t*, const uint64_t*, const float*, const float*, float*);
template void naive_crs_spmv_cuda_launcher<uint64_t, double>(ULL, const uint64_t*, const uint64_t*, const double*, const double*, double*);
// clang-format on

} // namespace SMAX::KERNELS::SPMV::CUDA
